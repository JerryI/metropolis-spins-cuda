#include "hip/hip_runtime.h"
/*
 * For a description of the algorithm and the terms used, please see the
 * documentation for this sample.
 *
 * Each work-item invocation of this kernel, calculates the position for 
 * one particle
 *
 * Work-items use local memory to reduce memory bandwidth and reuse of data
 */
inline __host__ __device__ float4 operator+(float4 a, float4 b)
{
    return make_float4(a.x + b.x, a.y + b.y, a.z + b.z,  a.w + b.w);
}

inline __host__ __device__ float4 operator-(float4 a, float4 b)
{
    return make_float4(a.x - b.x, a.y - b.y, a.z - b.z,  a.w - b.w);
}

inline __host__ __device__ float4 operator*(float4 a, float b)
{
    return make_float4(a.x * b, a.y  * b, a.z  * b,  a.w  * b);
}

inline __host__ __device__ void operator+=(float4 &a, float4 b)
{
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
    a.w += b.w;
}

__device__ int getGlobalId() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

__device__ int getLocalId() {
    return (threadIdx.y * blockDim.x) + threadIdx.x;
}

__device__ int getGroupId() {
    return blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
}

__device__ int getLocalSize() {
    return (blockDim.x * blockDim.y);
}


extern "C"
__global__ void calcFields(
    float4* spin,
    const float4* pos,

    const float4*  newSpin,

    const float Jxx,
    const float Jyy,
    const float Jxy,
    const float DIP,
    const float tresh,

    float4* partialField,

    const mint next,
    const mint prev
)

{
    __shared__ float4 localSums[1024];

    unsigned int tid = getLocalId();
    unsigned int bid = getGroupId();
    unsigned int localSize = getLocalSize();
    unsigned int gid = getGlobalId();

    //Write the previous
    if(gid == 0) {
        spin[prev] = newSpin[0];
    }

    float4 nextPos = pos[next];
    float4 privatepos = pos[gid];
    float4 r = privatepos - nextPos;

    float distSqr = r.x * r.x  +  r.y * r.y  +  r.z * r.z;

    if (distSqr < 0.1f) {
        localSums[tid] = make_float4(0.0,0.0,0.0,0.0);
    } else {
        float invDist = 1.0f / sqrt(distSqr);
        float invDistCube = invDist * invDist * invDist;  
        float invDistPenta = invDistCube * invDist * invDist;
        float4 privatefield; float4 privatespin = spin[gid];

        privatefield.x = privatespin.x * (invDistCube - 3 * invDistPenta * r.x * r.x) - privatespin.y * 3 * invDistPenta * r.x * r.y - privatespin.z * 3 * invDistPenta * r.x * r.z; 
        privatefield.y = privatespin.y * (invDistCube - 3 * invDistPenta * r.y * r.y) - privatespin.z * 3 * invDistPenta * r.y * r.z - privatespin.x * 3 * invDistPenta * r.y * r.x; 
        privatefield.z = privatespin.z * (invDistCube - 3 * invDistPenta * r.z * r.z) - privatespin.x * 3 * invDistPenta * r.z * r.x - privatespin.y * 3 * invDistPenta * r.z * r.y; 

        privatefield.x = privatefield.x * DIP;
        privatefield.y = privatefield.y * DIP;
        privatefield.z = privatefield.z * DIP;

        float J = Jxy;

        if (distSqr < tresh) {

            if (privatepos.w > 0.0f && nextPos.w > 0.0f) {
                J = Jyy;
            } else if (privatepos.w < 0.0f && nextPos.w < 0.0f) {
                J = Jxx;
            }                

            privatefield.x = privatefield.x + privatespin.x * J;
            privatefield.y = privatefield.y + privatespin.y * J;
            privatefield.z = privatefield.z + privatespin.z * J;
        }

        localSums[tid].x = privatefield.x;
        localSums[tid].y = privatefield.y;
        localSums[tid].z = privatefield.z;
    }

    // Loop for computing localSums : divide WorkGroup into 2 parts
    for (unsigned int stride = localSize/2; stride>0; stride /=2)
    {
        // Waiting for each 2x2 addition into given workgroup
        __syncthreads();

        // Add elements 2 by 2 between local_id and local_id + stride
        if (tid < stride)
            localSums[tid] += localSums[tid + stride];
    }

    __syncthreads();
    // Write result into partialSums[nWorkGroups]
    if (tid == 0) {
        partialField[bid].x = localSums[0].x;
        partialField[bid].y = localSums[0].y;
        partialField[bid].z = localSums[0].z;
    }

    if (gid == 0) {
        partialField[0].w = pos[next].w;
    }    

}


__global__ void calcMag(
    float4* spin,
    float4* partialField
)

{
    __shared__ float4 localSums[1024];

    unsigned int tid = getLocalId();
    unsigned int bid = getGroupId();
    unsigned int localSize = getLocalSize();
    unsigned int gid = getGlobalId();



    localSums[tid].x = 2*spin[gid].x;
    localSums[tid].y = 2*spin[gid].y;
    localSums[tid].z = 2*spin[gid].z;

    // Loop for computing localSums : divide WorkGroup into 2 parts
    for (unsigned int stride = localSize/2; stride>0; stride /=2)
    {
        // Waiting for each 2x2 addition into given workgroup
        __syncthreads();

        // Add elements 2 by 2 between local_id and local_id + stride
        if (tid < stride)
            localSums[tid] += localSums[tid + stride];
    }

    __syncthreads();
    // Write result into partialSums[nWorkGroups]
    if (tid == 0) {
        partialField[bid].x = localSums[0].x;
        partialField[bid].y = localSums[0].y;
        partialField[bid].z = localSums[0].z;
    } 

}

